#include "hip/hip_runtime.h"
#include "FloydWarshall.hpp"
#include "Graph/GraphWeight.hpp"
#include <tuple>
#include <limits>
#include <iomanip>
#include <iostream>
#include <random>
#include <chrono>

#include <thread>

#include <fstream>


using matrix_t = float; //matrix_t is like float

//TIME
using namespace std;
using namespace std::chrono;

//nV => numero di vettori del grafo
//nE => numero di archi nel grafo


const int BLOCK_SIZE_X = 32;
const int BLOCK_SIZE_Y = 32;
const float INF = std::numeric_limits<float>::infinity();

template<typename T> __global__ void floyd_warshall_kernel(T* matrix, int num_vertices, int k) {

    __shared__ float distyk[BLOCK_SIZE_Y];
    __shared__ float distkx[BLOCK_SIZE_X];

    unsigned int X = blockIdx.x * blockDim.x +threadIdx.x;  //colonne
    unsigned int Y = blockIdx.y * blockDim.y +threadIdx.y;  //righe

    if(X<num_vertices && Y<num_vertices){
        
        //float distyx = matrix[Y*num_vertices+X];

        if(Y % BLOCK_SIZE_Y == 0)
            distkx[X%BLOCK_SIZE_X] = matrix[k*num_vertices+X];
        
        if(X % BLOCK_SIZE_X == 0)
            distyk[Y%BLOCK_SIZE_Y] = matrix[Y*num_vertices+k];

        __syncthreads();
        if (distyk[Y%BLOCK_SIZE_Y] != INF &&
            distkx[X%BLOCK_SIZE_X] != INF &&
            distyk[Y%BLOCK_SIZE_Y] + distkx[X%BLOCK_SIZE_X] < matrix[Y*num_vertices + X])
                matrix[Y*num_vertices + X] = matrix[Y*num_vertices + k] + matrix[k*num_vertices + X];
    } 
}

__host__ int main(int argc, char* argv[]) {

    // se non trova due argomenti termina perchè manca l'input
    if (argc != 2)
        return EXIT_FAILURE;

    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    graph::GraphWeight<int, int, matrix_t> graph(graph::structure_prop::COO);   // arco da int A a int B di peso float P 
    graph.read(argv[1]);    //si copia da file i valori dati

    auto matrix = new matrix_t*[graph.nV()];    //usata per la parte di codice sequenziale
    const int nV = graph.nV();

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    for (int i = 0; i < graph.nV(); i++) {
        matrix[i] = new matrix_t[graph.nV()];
        std::fill(matrix[i], matrix[i] + graph.nV(), std::numeric_limits<matrix_t>::infinity());    //inizializza tutti gli archi a INF
    }
    for (int i = 0; i < graph.nE(); i++) {
        auto index = graph.coo_ptr()[i];
        matrix[std::get<0>(index)][std::get<1>(index)] = std::get<2>(index);
    }

    //--------------------------------------------------------------------------
    // HOST EXECUTIION

    float* h_matrix = new float[graph.nV()*graph.nV()];

    for (int i = 0; i < graph.nV(); i++)
        for(int j = 0; j < graph.nV(); j++)
            h_matrix[i*graph.nV() + j] = matrix[i][j];

    auto t1 = std::chrono::system_clock::now();
    //std::this_thread::sleep_for(seconds(5));
    floyd_warshall::floyd_warshall(matrix, graph.nV()); //codice sequenziale
    auto t2 = std::chrono::system_clock::now();
    long double duration = duration_cast<milliseconds>( t2 - t1 ).count();
    std::cout << "TIME CPU " << duration/1000 << '\n';

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    float *d_matrix;
    int dim_data = graph.nV()*graph.nV()*sizeof(float);
    hipMalloc(&d_matrix, dim_data);

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    hipMemcpy(d_matrix , h_matrix, dim_data, hipMemcpyHostToDevice);

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION

    dim3 block_size(BLOCK_SIZE_X,BLOCK_SIZE_Y);
    dim3 num_blocks( (graph.nV()+BLOCK_SIZE_X-1)/BLOCK_SIZE_X, (graph.nV()+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y);

    /// time start
    hipEvent_t startTimeCuda, stopTimeCuda;
    hipEventCreate(&startTimeCuda);
    hipEventCreate(&stopTimeCuda);
    hipEventRecord(startTimeCuda, 0);

    for(int k = 0; k < graph.nV(); k++)
        floyd_warshall_kernel<<< num_blocks, block_size >>>(d_matrix,nV, k);
    
    hipEventRecord(stopTimeCuda,0);
    hipEventSynchronize(stopTimeCuda);
    float msTime;
    hipEventElapsedTime(&msTime, startTimeCuda, stopTimeCuda);

    std::cout << "TIME GPU " << msTime/1000 << '\n';
    std::cout << "SPEEDUP " << (duration/1000)/(msTime/1000) << '\n';

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    hipMemcpy(h_matrix, d_matrix, dim_data, hipMemcpyDeviceToHost);

    // CREATE A FILE OF VALUES
    std::ofstream myfile;
    myfile.open ("../data.csv");
    for (int i = 0; i < graph.nV(); i++){
        for(int j = 0; j < graph.nV(); j++){
            myfile << h_matrix[i*graph.nV() + j] << ",";
        }
        myfile << '\n';
    }
    myfile.close();

    //--------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < graph.nV(); i++)
        for(int j = 0; j < graph.nV(); j++)
            if (h_matrix[i*graph.nV()+j] != matrix[i][j]) {
                std::cerr << "wrong result at: ("
                        << i << ", " << j << ")"
                        << "\nhost:   " << matrix[i][j]
                        << "\ndevice: " << h_matrix[i*graph.nV()+j] << "\n\n";
                hipDeviceReset();
                std::exit(EXIT_FAILURE);
            }

    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix;
    delete[] matrix;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    hipFree(d_matrix);

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
